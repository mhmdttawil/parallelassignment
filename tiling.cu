#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16  // Size of each tile

__global__ void tiled_matrix_mult(int *a, int *b, int *c, int n) {
    __shared__ int shared_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int shared_b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int sum = 0;

    for (int tile = 0; tile < (n + BLOCK_SIZE - 1) / BLOCK_SIZE; tile++) {
        // Load data into shared memory
        if (row < n && tile * BLOCK_SIZE + threadIdx.x < n) {
            shared_a[threadIdx.y][threadIdx.x] = a[row * n + tile * BLOCK_SIZE + threadIdx.x];
        } else {
            shared_a[threadIdx.y][threadIdx.x] = 0;
        }

        if (col < n && tile * BLOCK_SIZE + threadIdx.y < n) {
            shared_b[threadIdx.y][threadIdx.x] = b[(tile * BLOCK_SIZE + threadIdx.y) * n + col];
        } else {
            shared_b[threadIdx.y][threadIdx.x] = 0;
        }

        __syncthreads();  // Ensure all threads in the block have loaded the data

        // Perform the dot product within the tile
        for (int k = 0; k < BLOCK_SIZE; k++) {
            sum += shared_a[threadIdx.y][k] * shared_b[k][threadIdx.x];
        }

        __syncthreads();  // Ensure all computations in the tile are done before moving to the next tile
    }

    if (row < n && col < n) {
        c[row * n + col] = sum;
    }
}

int main(int argc, char const *argv[]) {
    int n;

    for (n = 64; n <= 8192; n *= 2) {
        int *h_a, *h_b, *h_c;
        hipHostMalloc((void **) &h_a, sizeof(int) * n * n, hipHostMallocDefault);
        hipHostMalloc((void **) &h_b, sizeof(int) * n * n, hipHostMallocDefault);
        hipHostMalloc((void **) &h_c, sizeof(int) * n * n, hipHostMallocDefault);

        // Initialize matrices A and B
        for (int i = 0; i < n; i++) {
            for (int j = 0; j < n; j++) {
                h_a[i * n + j] = 2;
                h_b[i * n + j] = 3;
            }
        }

        float tiled_gpu_elapsed_time_ms;

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        int *d_a, *d_b, *d_c;
        hipMalloc((void **) &d_a, sizeof(int) * n * n);
        hipMalloc((void **) &d_b, sizeof(int) * n * n);
        hipMalloc((void **) &d_c, sizeof(int) * n * n);

        hipMemcpy(d_a, h_a, sizeof(int) * n * n, hipMemcpyHostToDevice);
        hipMemcpy(d_b, h_b, sizeof(int) * n * n, hipMemcpyHostToDevice);

        unsigned int grid_rows = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
        unsigned int grid_cols = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
        dim3 dimGrid(grid_cols, grid_rows);
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

        hipEventRecord(start, 0);
        tiled_matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, n);

        hipDeviceSynchronize();  // Ensure the kernel completes before timing

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&tiled_gpu_elapsed_time_ms, start, stop);

        hipMemcpy(h_c, d_c, sizeof(int) * n * n, hipMemcpyDeviceToHost);

        printf("Time elapsed on tiled GPU matrix multiplication of %dx%d: %f ms.\n", n, n, tiled_gpu_elapsed_time_ms);

        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
        hipHostFree(h_a);
        hipHostFree(h_b);
        hipHostFree(h_c);
    }

    return 0;
}
